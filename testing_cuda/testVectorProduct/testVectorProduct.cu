#include "hip/hip_runtime.h"

// System includes
#include <stdio.h>
#include <assert.h>
#include <stdlib.h> // For rand(), RAND_MAX


// CUDA runtime
#include <hip/hip_runtime.h>

typedef float DATATYPE;

const int n = 4096;
const int threadnum = 1024;

__global__ void vector_dot_prod(DATATYPE *a, DATATYPE *b, DATATYPE *c, int n) {
	if (threadIdx.x == 0 && blockIdx.x == 0) {
		c[0] = 0.0;
	}
	
	__shared__ DATATYPE tmp[threadnum];
	const int tidx = threadIdx.x;
	const int bidx =  blockIdx.x;
	const int t_n = blockDim.x * gridDim.x;
	int tid = bidx * blockDim.x + tidx;
	double temp = 0.0;
	while (tid < n) {
		temp += a[tid]*b[tid];
		tid += t_n;
	}
	tmp[tidx] = temp;

	__syncthreads();

	int i = blockDim.x/2;
	while (i != 0) {
		if (tidx < i) {
			tmp[tidx] += tmp[tidx+i];
		}

		__syncthreads();

		i /= 2;
	}

	if (tidx == 0) {
		atomicAdd(c, tmp[0]);
	}
}


int calc_doct_prod(DATATYPE *a, DATATYPE *b, DATATYPE *c) {

	DATATYPE *d_a;
	DATATYPE *d_b;
	DATATYPE *d_c;

	// GPU memory alloc
	hipMalloc((void**)&d_a, sizeof(DATATYPE)*n);
	hipMalloc((void**)&d_b, sizeof(DATATYPE)*n);
	hipMalloc((void**)&d_c, sizeof(DATATYPE)*n);

	// GPU data transfer
	hipMemcpy(d_a, a, sizeof(DATATYPE)*n, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, sizeof(DATATYPE)*n, hipMemcpyHostToDevice);

	// call the GPU kernel
	vector_dot_prod<<<(int)ceil(n/threadnum),threadnum>>>(d_a, d_b, d_c, n);

	// result copy back to CPU
	hipMemcpy(c, d_c, sizeof(DATATYPE)*n, hipMemcpyDeviceToHost);

	// GPU memory free
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	return 0;
}



int calc_doct_prod_CPU(DATATYPE *a, DATATYPE *b, DATATYPE *c) {

    DATATYPE s = 0.0;

	for (int i = 0; i < n; i++) {
		s += a[i] * b[i];
	}
    c[0] = s;

	return 0;
}

int main(){

    srand(time(0));

	DATATYPE *a;
	DATATYPE *b;
	DATATYPE *c;

	a = (DATATYPE *)malloc(sizeof(DATATYPE)*n);
	b = (DATATYPE *)malloc(sizeof(DATATYPE)*n);
	c = (DATATYPE *)malloc(sizeof(DATATYPE)*n);

	for (int i = 0; i < n; i++) {
		a[i] = (DATATYPE)rand()/RAND_MAX;
		b[i] = (DATATYPE)rand()/RAND_MAX;
	}

	calc_doct_prod(a, b, c);
	// calc_doct_prod_CPU(a, b, c);
	
	printf("%f", c[0]);


	free(a);
	free(b);
	free(c);

	return 0;
}
